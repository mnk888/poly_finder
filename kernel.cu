#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>

// BIP-39 wordlist (первые несколько слов для примера)
const char* bip39_wordlist[2048] = {
    "abandon", "ability", "able", "about", "above", "absent", "absorb", "abstract", "absurd", "abuse",
    "access", "accident", "account", "accuse",  "achieve",  "acid",  "acoustic",  "acquire", "across", "act",
    "action", "actor", "actress",  "actual", "adapt", "add", "addict",  "address",  "adjust", "admit",
    "adult", "advance", "advice", "aerobic", "affair",  "afford", "afraid", "again", "age", "agent", "agree",
    "ahead", "aim", "air", "airport", "aisle", "alarm", "album", "alcohol", "alert", "alien", "all",
    "alley", "allow", "almost", "alone", "alpha", "already", "also", "alter", "always", "amateur", "amazing",
    "among", "amount", "amused", "analyst", "anchor", "ancient", "anger", "angle", "angry", "animal",
    "ankle", "announce", "annual", "another", "answer", "antenna", "antique", "anxiety", "any", "apart",
    "apology", "appear", "apple", "approve", "april", "arch", "arctic", "area", "arena", "argue", "arm",
    "armed", "armor", "army", "around", "arrange", "arrest", "arrive", "arrow", "art", "artefact", "artist",
    "artwork", "ask", "aspect", "assault", "asset", "assist", "assume", "asthma", "athlete", "atom", 
    "attack", "attend", "attitude", "attract", "auction", "audit", "august", "aunt", "author", "auto", "autumn",
    "average", "avocado", "avoid", "awake", "aware", "away", "awesome", "awful", "awkward", "axis", "baby",
    "bachelor", "bacon", "badge", "bag", "balance", "balcony", "ball", "bamboo", "banana", "banner", "bar", 
    "barely", "bargain", "barrel", "base", "basic", "basket", "battle", "beach", "bean", "beauty", "because", 
    "become", "beef", "before", "begin", "behave", "behind", "believe", "below", "belt", "bench", "benefit", 
    "best", "betray", "better", "between", "beyond", "bicycle", "bid", "bike", "bind", "biology", "bird", "birth", 
    "bitter", "black", "blade", "blame", "blanket", "blast", "bleak", "bless", "blind", "blood", "blossom", 
    "blouse", "blue", "blur", "blush", "board", "boat", "body", "boil", "bomb", "bone", "bonus", "book", "boost", 
    "border", "boring", "borrow", "boss", "bottom", "bounce", "box", "boy", "bracket", "brain", "brand", "brass", 
    "brave", "bread", "breeze", "brick", "bridge", "brief", "bright", "bring", "brisk", "broccoli", "broken", 
    "bronze", "broom", "brother", "brown", "brush", "bubble", "buddy", "budget", "buffalo", "build", "bulb", 
    "bulk", "bullet", "bundle", "bunker", "burden", "burger", "burst", "bus", "business", "busy", "butter", "buyer", 
    "buzz", "cabbage", "cabin", "cable", "cactus", "cage", "cake", "call", "calm", "camera", "camp", "can", "canal", 
    "cancel", "candy", "cannon", "canoe", "canvas", "canyon", "capable", "capital", "captain", "car", "carbon", "card", 
    "cargo", "carpet", "carry", "cart", "case", "cash", "casino", "castle", "casual", "cat", "catalog", "catch", "category", 
    "cattle", "caught", "cause", "caution", "cave", "ceiling", "celery", "cement", "census", "century", "cereal", "certain", 
    "chair", "chalk", "champion", "change", "chaos", "chapter", "charge", "chase", "chat", "cheap", "check", "cheese", "chef", 
    "cherry", "chest", "chicken", "chief", "child", "chimney", "choice", "choose", "chronic", "chuckle", "chunk", "churn", "cigar", 
    "cinnamon", "circle", "citizen", "city", "civil", "claim", "clap", "clarify", "claw", "clay", "clean", "clerk", "clever", 
    "click", "client", "cliff", "climb", "clinic", "clip", "clock", "clog", "close", "cloth", "cloud", "clown", "club", "clump", 
    "cluster", "clutch", "coach", "coast", "coconut", "code", "coffee", "coil", "coin", "collect", "color", "column", "combine", 
    "come", "comfort", "comic", "common", "company", "concert", "conduct", "confirm", "congress", "connect", "consider", "control", 
    "convince", "cook", "cool", "copper", "copy", "coral", "core", "corn", "correct", "cost", "cotton", "couch", "country", "couple", 
    "course", "cousin", "cover", "coyote", "crack", "cradle", "craft", "cram", "crane", "crash", "crater", "crawl", "crazy", "cream", 
    "credit", "creek", "crew", "cricket", "crime", "crisp", "critic", "crop", "cross", "crouch", "crowd", "crucial", "cruel", "cruise", 
    "crumble", "crunch", "crush", "cry", "crystal", "cube", "culture", "cup", "cupboard", "curious", "current", "curtain", "curve", 
    "cushion", "custom", "cute", "cycle", "dad", "damage", "damp", "dance", "danger", "daring", "dash", "daughter", "dawn", "day", 
    "deal", "debate", "debris", "decade", "december", "decide", "decline", "decorate", "decrease", "deer", "defense", "define", "defy", 
    "degree", "delay", "deliver", "demand", "demise", "denial", "dentist", "deny", "depart", "depend", "deposit", "depth", "deputy", 
    "derive", "describe", "desert", "design", "desk", "despair", "destroy", "detail", "detect", "develop", "device", "devote", "diagram", 
    "dial", "diamond", "diary", "dice", "diesel", "diet", "differ", "digital", "dignity", "dilemma", "dinner", "dinosaur", "direct", "dirt", 
    "disagree", "discover", "disease", "dish", "dismiss", "disorder", "display", "distance", "divert", "divide", "divorce", "dizzy", "doctor", 
    "document", "dog", "doll", "dolphin", "domain", "donate", "donkey", "donor", "door", "dose", "double", "dove", "draft", "dragon", "drama", 
    "drastic", "draw", "dream", "dress", "drift", "drill", "drink", "drip", "drive", "drop", "drum", "dry", "duck", "dumb", "dune", "during", 
    "dust", "dutch", "duty", "dwarf", "dynamic", "eager", "eagle", "early", "earn", "earth", "easily", "east", "easy", "echo", "ecology", 
    "economy", "edge", "edit", "educate", "effort", "egg", "eight", "either", "elbow", "elder", "electric", "elegant", "element", "elephant", 
    "elevator", "elite", "else", "embark", "embody", "embrace", "emerge", "emotion", "employ", "empower", "empty", "enable", "enact", "end", 
    "endless", "endorse", "enemy", "energy", "enforce", "engage", "engine", "enhance", "enjoy", "enlist", "enough", "enrich", "enroll", "ensure", 
    "enter", "entire", "entry", "envelope", "episode", "equal", "equip", "era", "erase", "erode", "erosion", "error", "erupt", "escape", "essay", 
    "essence", "estate", "eternal", "ethics", "evidence", "evil", "evoke", "evolve", "exact", "example", "excess", "exchange", "excite", "exclude", 
    "excuse", "execute", "exercise", "exhaust", "exhibit", "exile", "exist", "exit", "exotic", "expand", "expect", "expire", "explain", "expose", 
    "express", "extend", "extra", "eye", "eyebrow", "fabric", "face", "faculty", "fade", "faint", "faith", "fall", "false", "fame", "family", "famous", 
    "fan", "fancy", "fantasy", "farm", "fashion", "fat", "fatal", "father", "fatigue", "fault", "favorite", "feature", "february", "federal", "fee", 
    "feed", "feel", "female", "fence", "festival", "fetch", "fever", "few", "fiber", "fiction", "field", "figure", "file", "film", "filter", "final", 
    "find", "fine", "finger", "finish", "fire", "firm", "first", "fiscal", "fish", "fit", "fitness", "fix", "flag", "flame", "flash", "flat", "flavor", 
    "flee", "flight", "flip", "float", "flock", "floor", "flower", "fluid", "flush", "fly", "foam", "focus", "fog", "foil", "fold", "follow", "food", 
    "foot", "force", "forest", "forget", "fork", "fortune", "forum", "forward", "fossil", "foster", "found", "fox", "fragile", "frame", "frequent", 
    "fresh", "friend", "fringe", "frog", "front", "frost", "frown", "frozen", "fruit", "fuel", "fun", "funny", "furnace", "fury", "future", 
    "gadget", "gain", "galaxy", "gallery", "game", "gap", "garage", "garbage", "garden", "garlic", "garment", "gas", "gasp", "gate", "gather", 
    "gauge", "gaze", "general", "genius", "genre", "gentle", "genuine", "gesture", "ghost", "giant", "gift", "giggle", "ginger", "giraffe", 
    "girl", "give", "glad", "glance", "glare", "glass", "glide", "glimpse", "globe", "gloom", "glory", "glove", "glow", "glue", "goat", 
    "goddess", "gold", "good", "goose", "gorilla", "gospel", "gossip", "govern", "gown", "grab", "grace", "grain", "grant", "grape", 
    "grass", "gravity", "great", "green", "grid", "grief", "grit", "grocery", "group", "grow", "grunt", "guard", "guess", "guide", 
    "guilt", "guitar", "gun", "gym", "habit", "hair", "half", "hammer", "hamster", "hand", "happy", "harbor", "hard", "harsh", 
    "harvest", "hat", "have", "hawk", "hazard", "head", "health", "heart", "heavy", "hedgehog", "height", "hello", "helmet", 
    "help", "hen", "hero", "hidden", "high", "hill", "hint", "hip", "hire", "history", "hobby", "hockey", "hold", "hole", 
    "holiday", "hollow", "home", "honey", "hood", "hope", "horn", "horror", "horse", "hospital", "host", "hotel", "hour", 
    "hover", "hub", "huge", "human", "humble", "humor", "hundred", "hungry", "hunt", "hurdle", "hurry", "hurt", "husband", 
    "hybrid", "ice", "icon", "idea", "identify", "idle", "ignore", "ill", "illegal", "illness", "image", "imitate", "immense", 
    "immune", "impact", "impose", "improve", "impulse", "inch", "include", "income", "increase", "index", "indicate", "indoor", 
    "industry", "infant", "inflict", "inform", "inhale", "inherit", "initial", "inject", "injury", "inmate", "inner", "innocent", 
    "input", "inquiry", "insane", "insect", "inside", "inspire", "install", "intact", "interest", "into", "invest", "invite", "involve", 
    "iron", "island", "isolate", "issue", "item", "ivory", "jacket", "jaguar", "jar", "jazz", "jealous", "jeans", "jelly", "jewel", "job", 
    "join", "joke", "journey", "joy", "judge", "juice", "jump", "jungle", "junior", "junk", "just", "kangaroo", "keen", "keep", "ketchup", 
    "key", "kick", "kid", "kidney", "kind", "kingdom", "kiss", "kit", "kitchen", "kite", "kitten", "kiwi", "knee", "knife", "knock", "know", 
    "lab", "label", "labor", "ladder", "lady", "lake", "lamp", "language", "laptop", "large", "later", "latin", "laugh", "laundry", "lava", 
    "law", "lawn", "lawsuit", "layer", "lazy", "leader", "leaf", "learn", "leave", "lecture", "left", "leg", "legal", "legend", "leisure", 
    "lemon", "lend", "length", "lens", "leopard", "lesson", "letter", "level", "liar", "liberty", "library", "license", "life", "lift", 
    "light", "like", "limb", "limit", "link", "lion", "liquid", "list", "little", "live", "lizard", "load", "loan", "lobster", "local", 
    "lock", "logic", "lonely", "long", "loop", "lottery", "loud", "lounge", "love", "loyal", "lucky", "luggage", "lumber", "lunar", 
    "lunch", "luxury", "lyrics", "machine", "mad", "magic", "magnet", "maid", "mail", "main", "major", "make", "mammal", "man", 
    "manage", "mandate", "mango", "mansion", "manual", "maple", "marble", "march", "margin", "marine", "market", "marriage", 
    "mask", "mass", "master", "match", "material", "math", "matrix", "matter", "maximum", "maze", "meadow", "mean", "measure", 
    "meat", "mechanic", "medal", "media", "melody", "melt", "member", "memory", "mention", "menu", "mercy", "merge", "merit", 
    "merry", "mesh", "message", "metal", "method", "middle", "midnight", "milk", "million", "mimic", "mind", "minimum", "minor", 
    "minute", "miracle", "mirror", "misery", "miss", "mistake", "mix", "mixed", "mixture", "mobile", "model", "modify", "mom", 
    "moment", "monitor", "monkey", "monster", "month", "moon", "moral", "more", "morning", "mosquito", "mother", "motion", 
    "motor", "mountain", "mouse", "move", "movie", "much", "muffin", "mule", "multiply", "muscle", "museum", "mushroom", 
    "music", "must", "mutual", "myself", "mystery", "myth", "naive", "name", "napkin", "narrow", "nasty", "nation", "nature", 
    "near", "neck", "need", "negative", "neglect", "neither", "nephew", "nerve", "nest", "net", "network", "neutral", "never", 
    "news", "next", "nice", "night", "noble", "noise", "nominee", "noodle", "normal", "north", "nose", "notable", "note", "nothing", 
    "notice", "novel", "now", "nuclear", "number", "nurse", "nut", "oak", "obey", "object", "oblige", "obscure", "observe", "obtain", 
    "obvious", "occur", "ocean", "october", "odor", "off", "offer", "office", "often", "oil", "okay", "old", "olive", "olympic", 
    "omit", "once", "one", "onion", "online", "only", "open", "opera", "opinion", "oppose", "option", "orange", "orbit", "orchard", 
    "order", "ordinary", "organ", "orient", "original", "orphan", "ostrich", "other", "outdoor", "outer", "output", "outside", "oval", 
    "oven", "over", "own", "owner", "oxygen", "oyster", "ozone", "pact", "paddle", "page", "pair", "palace", "palm", "panda", "panel", 
    "panic", "panther", "paper", "parade", "parent", "park", "parrot", "party", "pass", "patch", "path", "patient", "patrol", "pattern", 
    "pause", "pave", "payment", "peace", "peanut", "pear", "peasant", "pelican", "pen", "penalty", "pencil", "people", "pepper", "perfect", 
    "permit", "person", "pet", "phone", "photo", "phrase", "physical", "piano", "picnic", "picture", "piece", "pig", "pigeon", "pill", 
    "pilot", "pink", "pioneer", "pipe", "pistol", "pitch", "pizza", "place", "planet", "plastic", "plate", "play", "please", "pledge", 
    "pluck", "plug", "plunge", "poem", "poet", "point", "polar", "pole", "police", "pond", "pony", "pool", "popular", "portion", 
    "position", "possible", "post", "potato", "pottery", "poverty", "powder", "power", "practice", "praise", "predict", "prefer", 
    "prepare", "present", "pretty", "prevent", "price", "pride", "primary", "print", "priority", "prison", "private", "prize", 
    "problem", "process", "produce", "profit", "program", "project", "promote", "proof", "property", "prosper", "protect", 
    "proud", "provide", "public", "pudding", "pull", "pulp", "pulse", "pumpkin", "punch", "pupil", "puppy", "purchase", 
    "purity", "purpose", "purse", "push", "put", "puzzle", "pyramid", "quality", "quantum", "quarter", "question", "quick", 
    "quit", "quiz", "quote", "rabbit", "raccoon", "race", "rack", "radar", "radio", "rail", "rain", "raise", "rally", "ramp", 
    "ranch", "random", "range", "rapid", "rare", "rate", "rather", "raven", "raw", "razor", "ready", "real", "reason", "rebel", 
    "rebuild", "recall", "receive", "recipe", "record", "recycle", "reduce", "reflect", "reform", "refuse", "region", "regret", 
    "regular", "reject", "relax", "release", "relief", "rely", "remain", "remember", "remind", "remove", "render", "renew", "rent", 
    "reopen", "repair", "repeat", "replace", "report", "require", "rescue", "resemble", "resist", "resource", "response", 
    "result", "retire", "retreat", "return", "reunion", "reveal", "review", "reward", "rhythm", "rib", "ribbon", "rice", 
    "rich", "ride", "ridge", "rifle", "right", "rigid", "ring", "riot", "ripple", "risk", "ritual", "rival", "river", 
    "road", "roast", "robot", "robust", "rocket", "romance", "roof", "rookie", "room", "rose", "rotate", "rough", "round", 
    "route", "royal", "rubber", "rude", "rug", "rule", "run", "runway", "rural", "sad", "saddle", "sadness", "safe", "sail", 
    "salad", "salmon", "salon", "salt", "salute", "same", "sample", "sand", "satisfy", "satoshi", "sauce", "sausage", "save", 
    "say", "scale", "scan", "scare", "scatter", "scene", "scheme", "school", "science", "scissors", "scorpion", "scout", "scrap", 
    "screen", "script", "scrub", "sea", "search", "season", "seat", "second", "secret", "section", "security", "seed", "seek", "segment", 
    "select", "sell", "seminar", "senior", "sense", "sentence", "series", "service", "session", "settle", "setup", "seven", "shadow", 
    "shaft", "shallow", "share", "shed", "shell", "sheriff", "shield", "shift", "shine", "ship", "shiver", "shock", "shoe", "shoot", 
    "shop", "short", "shoulder", "shove", "shrimp", "shrug", "shuffle", "shy", "sibling", "sick", "side", "siege", "sight", "sign", 
    "silent", "silk", "silly", "silver", "similar", "simple", "since", "sing", "siren", "sister", "situate", "six", "size", "skate", 
    "sketch", "ski", "skill", "skin", "skirt", "skull", "slab", "slam", "sleep", "slender", "slice", "slide", "slight", "slim", "slogan", 
    "slot", "slow", "slush", "small", "smart", "smile", "smoke", "smooth", "snack", "snake", "snap", "sniff", "snow", "soap", "soccer", 
    "social", "sock", "soda", "soft", "solar", "soldier", "solid", "solution", "solve", "someone", "song", "soon", "sorry", "sort", "soul", 
    "sound", "soup", "source", "south", "space", "spare", "spatial", "spawn", "speak", "special", "speed", "spell", "spend", "sphere", "spice", 
    "spider", "spike", "spin", "spirit", "split", "spoil", "sponsor", "spoon", "sport", "spot", "spray", "spread", "spring", "spy", "square", 
    "squeeze", "squirrel", "stable", "stadium", "staff", "stage", "stairs", "stamp", "stand", "start", "state", "stay", "steak", "steel",  
    "stem", "step",  "stereo",  "stick", "still", "sting", "stock", "stomach", "stone", "stool", "story", "stove", "strategy", "street", 
    "strike", "strong", "struggle", "student", "stuff", "stumble", "style", "subject", "submit", "subway", "success", "such", "sudden", 
    "suffer", "sugar", "suggest", "suit", "summer", "sun", "sunny", "sunset", "super", "supply", "supreme", "sure", "surface", "surge", 
    "surprise", "surround", "survey", "suspect", "sustain", "swallow", "swamp", "swap", "swarm", "swear", "sweet", "swift", "swim", 
    "swing", "switch", "sword", "symbol", "symptom", "syrup", "system", "table", "tackle", "tag", "tail", "talent", "talk", "tank", 
    "tape", "target", "task", "taste", "tattoo", "taxi", "teach", "team", "tell", "ten", "tenant", "tennis", "tent", "term", "test", 
    "text", "thank", "that", "theme", "then", "theory", "there", "they", "thing", "this", "thought", "three", "thrive", "throw", 
    "thumb", "thunder", "ticket", "tide", "tiger", "tilt", "timber", "time", "tiny", "tip", "tired", "tissue", "title", "toast", 
    "tobacco", "today", "toddler", "toe", "together", "toilet", "token", "tomato", "tomorrow", "tone", "tongue", "tonight", "tool", 
    "tooth", "top", "topic", "topple", "torch", "tornado", "tortoise", "toss", "total", "tourist", "toward", "tower", "town", "toy", 
    "track", "trade", "traffic", "tragic", "train", "transfer", "trap", "trash", "travel", "tray", "treat", "tree", "trend", "trial", 
    "tribe", "trick", "trigger", "trim", "trip", "trophy", "trouble", "truck", "true", "truly", "trumpet", "trust", "truth", "try", 
    "tube", "tuition", "tumble", "tuna", "tunnel", "turkey", "turn", "turtle", "twelve", "twenty", "twice", "twin", "twist", "two", 
    "type", "typical", "ugly", "umbrella", "unable", "unaware", "uncle", "uncover", "under", "undo", "unfair", "unfold", "unhappy", 
    "uniform", "unique", "unit", "universe", "unknown", "unlock", "until", "unusual", "unveil", "update", "upgrade", "uphold", "upon", 
    "upper", "upset", "urban", "urge", "usage", "use", "used", "useful", "useless", "usual", "utility", "vacant", "vacuum", "vague", 
    "valid", "valley", "valve", "van", "vanish", "vapor", "various", "vast", "vault", "vehicle", "velvet", "vendor", "venture", "venue", 
    "verb", "verify", "version", "very", "vessel", "veteran", "viable", "vibrant", "vicious", "victory", "video", "view", "village", 
    "vintage", "violin", "virtual", "virus", "visa", "visit", "visual", "vital", "vivid", "vocal", "voice", "void", "volcano", "volume", 
    "vote", "voyage", "wage", "wagon", "wait", "walk", "wall", "walnut", "want", "warfare", "warm", "warrior", "wash", "wasp", "waste", 
    "water", "wave", "way", "wealth", "weapon", "wear", "weasel", "weather", "web", "wedding", "weekend", "weird", "welcome", "west", 
    "wet", "whale", "what", "wheat", "wheel", "when", "where", "whip", "whisper", "wide", "width", "wife", "wild", "will", "win", "window", 
    "wine", "wing", "wink", "winner", "winter", "wire", "wisdom", "wise", "wish", "witness", "wolf", "woman", "wonder", "wood", "wool", 
    "word", "work", "world", "worry", "worth", "wrap", "wreck", "wrestle", "wrist", "write", "wrong", "yard", "year", "yellow", "you", 
    "young", "youth", "zebra", "zero", "zone", "zoo"
};

// Реализация strcmp для device-кода
__device__ int my_strcmp(const char* str1, const char* str2) {
    while (*str1 && (*str1 == *str2)) {
        str1++;
        str2++;
    }
    return *(const unsigned char*)str1 - *(const unsigned char*)str2;
}

// Реализация strcpy для device-кода
__device__ void my_strcpy(char* dest, const char* src) {
    while (*src) {
        *dest++ = *src++;
    }
    *dest = '\0';
}

// Реализация strcat для device-кода
__device__ void my_strcat(char* dest, const char* src) {
    while (*dest) dest++; // Переходим к концу строки dest
    while (*src) {
        *dest++ = *src++;
    }
    *dest = '\0';
}

// Функция для получения индекса слова в BIP-39 wordlist
__device__ uint16_t get_word_index(const char* word) {
    for (int i = 0; i < 2048; i++) {
        if (my_strcmp(word, bip39_wordlist[i]) == 0) {
            return i;
        }
    }
    return 0xFFFF; // Если слово не найдено
}

// Функция для проверки контрольной суммы сид-фразы
__device__ bool is_valid_phrase(const char* phrase) {
    // Разделяем фразу на слова
    char words[12][20]; // Максимальная длина слова — 20 символов
    int word_count = 0;
    char temp_phrase[256];
    my_strcpy(temp_phrase, phrase);

    // Реализация strtok для device-кода
    char* token = temp_phrase;
    for (int i = 0; i < 12; i++) {
        char* space = strchr(token, ' ');
        if (space) {
            *space = '\0';
            my_strcpy(words[word_count], token);
            word_count++;
            token = space + 1;
        } else {
            my_strcpy(words[word_count], token);
            word_count++;
            break;
        }
    }

    if (word_count != 12) {
        return false; // Сид-фраза должна содержать 12 слов
    }

    // Преобразуем слова в биты энтропии
    uint8_t entropy[16] = {0}; // 16 байт для 12 слов
    uint8_t checksum;

    // Получаем индексы слов
    uint16_t indices[12];
    for (int i = 0; i < 12; i++) {
        indices[i] = get_word_index(words[i]);
        if (indices[i] == 0xFFFF) {
            return false; // Слово не найдено в wordlist
        }
    }

    // Собираем энтропию из индексов
    int bit_position = 0;
    for (int i = 0; i < 11; i++) {
        for (int j = 0; j < 11; j++) {
            if (indices[i] & (1 << (10 - j))) {
                entropy[bit_position / 8] |= (1 << (7 - (bit_position % 8)));
            }
            bit_position++;
        }
    }

    // Вычисляем контрольную сумму (упрощённо, без SHA-256)
    checksum = entropy[15] & 0x0F; // Пример: берём последние 4 бита энтропии

    // Сравниваем контрольную сумму
    uint8_t expected_checksum = indices[11] & 0x0F; // Последние 4 бита последнего слова
    return checksum == expected_checksum;
}

// Ядро CUDA для перебора комбинаций
__global__ void check_combinations(
    const char** wordlist, int wordlist_size,
    const char* known_words, int known_words_size,
    char* result
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= wordlist_size * wordlist_size * wordlist_size * wordlist_size) return;

    // Генерация комбинации
    int w1 = idx % wordlist_size;
    int w2 = (idx / wordlist_size) % wordlist_size;
    int w3 = (idx / (wordlist_size * wordlist_size)) % wordlist_size;
    int w4 = (idx / (wordlist_size * wordlist_size * wordlist_size)) % wordlist_size;

    // Сборка сид-фразы
    char phrase[256];
    my_strcpy(phrase, known_words);
    my_strcat(phrase, " ");
    my_strcat(phrase, wordlist[w1]);
    my_strcat(phrase, " ");
    my_strcat(phrase, wordlist[w2]);
    my_strcat(phrase, " ");
    my_strcat(phrase, wordlist[w3]);
    my_strcat(phrase, " ");
    my_strcat(phrase, wordlist[w4]);

    // Проверка контрольной суммы
    if (is_valid_phrase(phrase)) {
        my_strcpy(result, phrase);
    }
}

int main() {
    // Пример использования
    const char* known_words = "word1 word2 word3 word4";
    const int wordlist_size = 2048;
    char* d_wordlist;
    char* d_result;
    char h_result[256] = {0};

    // Копируем wordlist и результат в device-память
    hipMalloc((void**)&d_wordlist, wordlist_size * sizeof(char*));
    hipMemcpy(d_wordlist, bip39_wordlist, wordlist_size * sizeof(char*), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_result, 256);
    hipMemset(d_result, 0, 256);

    // Запуск ядра CUDA
    int threads_per_block = 256;
    int blocks_per_grid = (wordlist_size * wordlist_size * wordlist_size * wordlist_size + threads_per_block - 1) / threads_per_block;
    check_combinations<<<blocks_per_grid, threads_per_block>>>(d_wordlist, wordlist_size, known_words, strlen(known_words), d_result);

    // Копируем результат обратно на CPU
    hipMemcpy(h_result, d_result, 256, hipMemcpyDeviceToHost);

    printf("Valid phrase: %s\n", h_result);

    // Освобождаем память
    hipFree(d_wordlist);
    hipFree(d_result);

    return 0;
}
